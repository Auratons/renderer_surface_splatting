#include "hip/hip_runtime.h"
#include <string>
#include <thread>

#include <boost/archive/text_oarchive.hpp>
#include <boost/serialization/vector.hpp>
#include <CLI/App.hpp>
#include <CLI/Formatter.hpp>  // Even thought seems unused it's needed
#include <CLI/Config.hpp>  // Even thought seems unused it's needed
#include <glm/glm.hpp>
#include <thrust/device_vector.h>

#include "ply_loader.hpp"

using namespace std;

int main(int argc, char** argv) {
  string pcd_path;
  CLI::App args{"Serializer for radii"};
  auto file = args.add_option("-f,--file", pcd_path, "Path to pointcloud to process");
  file->required();
  CLI11_PARSE(args, argc, argv);

  std::vector<glm::vec3> vertices_host, normals_host;
  std::vector<std::array<unsigned int, 3>> faces_host, colors_host;

  load_ply<glm::vec3>(pcd_path, vertices_host, normals_host, faces_host, colors_host);

//  std::vector<float3> vertices_host_f3(vertices_host.size());
//  std::transform(vertices_host.begin(), vertices_host.end(), vertices_host_f3.begin(), [](auto pt){ return make_float3(pt[0], pt[1], pt[2]); });

//  auto vertices = thrust::device_vector<float3>(vertices_host_f3.begin(), vertices_host_f3.end());
  auto vertices = thrust::device_vector<glm::vec3>(vertices_host.begin(), vertices_host.end());


//  auto radii = thrust::device_vector<float>(vertices_host.size());
//  std::vector<std::thread> threads(std::thread::hardware_concurrency());
//
//  for (std::size_t i(0); i < threads.size(); ++i)
//  {
//    std::size_t b = i * vertices.size() / threads.size();
//    std::size_t e = (i + 1) * vertices.size() / threads.size();
//
//    threads[i] = std::thread([b, e, &vertices, &radii]() {
//      //brute force
//      for (std::size_t i = b; i < e; ++i) {
//        auto tmp = thrust::device_vector<float>(vertices.size());
//        auto op = [vertices_begin = vertices.data().get(), current_target = i] __device__ (auto neighbor){
//          auto target_center = *(vertices_begin + current_target);
//          auto d = glm::length(target_center - neighbor);
//          return (d < 0.0005) ? 10.0f : d;  // There is always zero for the same point, this is workaround to get nn.
//        };
//        thrust::transform(vertices.begin(), vertices.end(), tmp.begin(), op);
//        radii[i] = *thrust::min_element(tmp.begin(), tmp.end());
//      }
//    });
//  }
//
//  for (auto& t : threads) { t.join(); }

  //brute force                                                                                                                                                                                                                               
  auto radii = thrust::device_vector<float>(vertices_host.size());
  auto tmp = thrust::device_vector<float>(vertices_host.size());
  for (size_t i = 0; i < vertices.size(); ++i) {
    auto op = [vertices_begin = vertices.data().get(), current_target = i] __device__ (auto neighbor){
        auto target_center = *(vertices_begin + current_target);
        auto d = glm::length(target_center - neighbor);
        return (d < 0.0005) ? 10.0f : d;  // There is always zero for the same point, this is workaround to get nn.
    };
    thrust::transform(vertices.begin(), vertices.end(), tmp.begin(), op);
    radii[i] = *thrust::min_element(tmp.begin(), tmp.end());
  }
  std::vector<float> radii_host(radii.begin(), radii.end());

  std::ofstream ofs(pcd_path + ".radii");
  boost::archive::text_oarchive oa(ofs);
  // write class instance to archive
  oa & radii_host;
}